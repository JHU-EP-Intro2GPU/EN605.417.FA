/**
 **********************************************************************
 ** Copyright (C) 1990, RSA Data Security, Inc. All rights reserved. **
 **                                                                  **
 ** License to copy and use this software is granted provided that   **
 ** it is identified as the "RSA Data Security, Inc. MD5 Message     **
 ** Digest Algorithm" in all material mentioning or referencing this **
 ** software or this function.                                       **
 **                                                                  **
 ** License is also granted to make and use derivative works         **
 ** provided that such works are identified as "derived from the RSA **
 ** Data Security, Inc. MD5 Message Digest Algorithm" in all         **
 ** material mentioning or referencing the derived work.             **
 **                                                                  **
 ** RSA Data Security, Inc. makes no representations concerning      **
 ** either the merchantability of this software or the suitability   **
 ** of this software for any particular purpose.  It is provided "as **
 ** is" without express or implied warranty of any kind.             **
 **                                                                  **
 ** These notices must be retained in any copies of any part of this **
 ** documentation and/or software.                                   **
 **********************************************************************
 */

#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <stdint.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/device_functions.h>

/* F, G and H are basic MD5 functions: selection, majority, parity */
#define F(x, y, z) (((x) & (y)) | ((~x) & (z)))
#define G(x, y, z) (((x) & (z)) | ((y) & (~z)))
#define H(x, y, z) ((x) ^ (y) ^ (z))
#define I(x, y, z) ((y) ^ ((x) | (~z)))

/* ROTATE_LEFT rotates x left n bits */
#define ROTATE_LEFT(x, n) (((x) << (n)) | ((x) >> (32-(n))))

/* FF, GG, HH, and II transformations for rounds 1, 2, 3, and 4 */
/* Rotation is separate from addition to prevent recomputation */
#define FFF(a, b, c, d, x, s, ac) \
  {(a) += F ((b), (c), (d)) + (x) + (uint32_t)(ac); \
    (a) = ROTATE_LEFT ((a), (s)); \
    (a) += (b); \
  }
#define GGG(a, b, c, d, x, s, ac) \
  {(a) += G ((b), (c), (d)) + (x) + (uint32_t)(ac); \
    (a) = ROTATE_LEFT ((a), (s)); \
    (a) += (b); \
  }
#define HHH(a, b, c, d, x, s, ac) \
  {(a) += H ((b), (c), (d)) + (x) + (uint32_t)(ac); \
    (a) = ROTATE_LEFT ((a), (s)); \
    (a) += (b); \
  }
#define III(a, b, c, d, x, s, ac) \
  {(a) += I ((b), (c), (d)) + (x) + (uint32_t)(ac); \
    (a) = ROTATE_LEFT ((a), (s)); \
    (a) += (b); \
  }

__device__ inline void md5Hash(unsigned char* data, uint32_t length, uint32_t *a1, uint32_t *b1, uint32_t *c1, uint32_t *d1){
  const uint32_t a0 = 0x67452301;
  const uint32_t b0 = 0xEFCDAB89;
  const uint32_t c0 = 0x98BADCFE;
  const uint32_t d0 = 0x10325476;

  uint32_t a = 0;
  uint32_t b = 0;
  uint32_t c = 0;
  uint32_t d = 0;

  uint32_t vals[14] = {0,0,0,0,0,0,0,0,0,0,0,0,0,0};

  int i = 0;
  for(i=0; i < length; i++){
    vals[i / 4] |= data[i] << ((i % 4) * 8);
  }
  
  vals[i / 4] |= 0x80 << ((i % 4) * 8);

  uint32_t bitlen = length * 8;

  #define in0  (vals[0])//x
  #define in1  (vals[1])//y
  #define in2  (vals[2])//z
  #define in3  (vals[3])
  #define in4  (vals[4])
  #define in5  (vals[5])
  #define in6  (vals[6])
  #define in7  (vals[7])
  #define in8  (vals[8])
  #define in9  (vals[9])
  #define in10 (vals[10])
  #define in11 (vals[11])
  #define in12 (vals[12])
  #define in13 (vals[13])
  #define in14 (bitlen) //w = bit length
  #define in15 (0)

  //Initialize hash value for this chunk:
  a = a0;
  b = b0;
  c = c0;
  d = d0;

  /* Round 1 */
  #define S11 7
  #define S12 12
  #define S13 17
  #define S14 22
  FFF ( a, b, c, d, in0,  S11, 3614090360); /* 1 */
  FFF ( d, a, b, c, in1,  S12, 3905402710); /* 2 */
  FFF ( c, d, a, b, in2,  S13,  606105819); /* 3 */
  FFF ( b, c, d, a, in3,  S14, 3250441966); /* 4 */
  FFF ( a, b, c, d, in4,  S11, 4118548399); /* 5 */
  FFF ( d, a, b, c, in5,  S12, 1200080426); /* 6 */
  FFF ( c, d, a, b, in6,  S13, 2821735955); /* 7 */
  FFF ( b, c, d, a, in7,  S14, 4249261313); /* 8 */
  FFF ( a, b, c, d, in8,  S11, 1770035416); /* 9 */
  FFF ( d, a, b, c, in9,  S12, 2336552879); /* 10 */
  FFF ( c, d, a, b, in10, S13, 4294925233); /* 11 */
  FFF ( b, c, d, a, in11, S14, 2304563134); /* 12 */
  FFF ( a, b, c, d, in12, S11, 1804603682); /* 13 */
  FFF ( d, a, b, c, in13, S12, 4254626195); /* 14 */
  FFF ( c, d, a, b, in14, S13, 2792965006); /* 15 */
  FFF ( b, c, d, a, in15, S14, 1236535329); /* 16 */

  /* Round 2 */
  #define S21 5
  #define S22 9
  #define S23 14
  #define S24 20
  GGG ( a, b, c, d, in1, S21, 4129170786); /* 17 */
  GGG ( d, a, b, c, in6, S22, 3225465664); /* 18 */
  GGG ( c, d, a, b, in11, S23,  643717713); /* 19 */
  GGG ( b, c, d, a, in0, S24, 3921069994); /* 20 */
  GGG ( a, b, c, d, in5, S21, 3593408605); /* 21 */
  GGG ( d, a, b, c, in10, S22,   38016083); /* 22 */
  GGG ( c, d, a, b, in15, S23, 3634488961); /* 23 */
  GGG ( b, c, d, a, in4, S24, 3889429448); /* 24 */
  GGG ( a, b, c, d, in9, S21,  568446438); /* 25 */
  GGG ( d, a, b, c, in14, S22, 3275163606); /* 26 */
  GGG ( c, d, a, b, in3, S23, 4107603335); /* 27 */
  GGG ( b, c, d, a, in8, S24, 1163531501); /* 28 */
  GGG ( a, b, c, d, in13, S21, 2850285829); /* 29 */
  GGG ( d, a, b, c, in2, S22, 4243563512); /* 30 */
  GGG ( c, d, a, b, in7, S23, 1735328473); /* 31 */
  GGG ( b, c, d, a, in12, S24, 2368359562); /* 32 */

  /* Round 3 */
  #define S31 4
  #define S32 11
  #define S33 16
  #define S34 23
  HHH ( a, b, c, d, in5, S31, 4294588738); /* 33 */
  HHH ( d, a, b, c, in8, S32, 2272392833); /* 34 */
  HHH ( c, d, a, b, in11, S33, 1839030562); /* 35 */
  HHH ( b, c, d, a, in14, S34, 4259657740); /* 36 */
  HHH ( a, b, c, d, in1, S31, 2763975236); /* 37 */
  HHH ( d, a, b, c, in4, S32, 1272893353); /* 38 */
  HHH ( c, d, a, b, in7, S33, 4139469664); /* 39 */
  HHH ( b, c, d, a, in10, S34, 3200236656); /* 40 */
  HHH ( a, b, c, d, in13, S31,  681279174); /* 41 */
  HHH ( d, a, b, c, in0, S32, 3936430074); /* 42 */
  HHH ( c, d, a, b, in3, S33, 3572445317); /* 43 */
  HHH ( b, c, d, a, in6, S34,   76029189); /* 44 */
  HHH ( a, b, c, d, in9, S31, 3654602809); /* 45 */
  HHH ( d, a, b, c, in12, S32, 3873151461); /* 46 */
  HHH ( c, d, a, b, in15, S33,  530742520); /* 47 */
  HHH ( b, c, d, a, in2, S34, 3299628645); /* 48 */

  /* Round 4 */
  #define S41 6
  #define S42 10
  #define S43 15
  #define S44 21
  III ( a, b, c, d, in0, S41, 4096336452); /* 49 */
  III ( d, a, b, c, in7, S42, 1126891415); /* 50 */
  III ( c, d, a, b, in14, S43, 2878612391); /* 51 */
  III ( b, c, d, a, in5, S44, 4237533241); /* 52 */
  III ( a, b, c, d, in12, S41, 1700485571); /* 53 */
  III ( d, a, b, c, in3, S42, 2399980690); /* 54 */
  III ( c, d, a, b, in10, S43, 4293915773); /* 55 */
  III ( b, c, d, a, in1, S44, 2240044497); /* 56 */
  III ( a, b, c, d, in8, S41, 1873313359); /* 57 */
  III ( d, a, b, c, in15, S42, 4264355552); /* 58 */
  III ( c, d, a, b, in6, S43, 2734768916); /* 59 */
  III ( b, c, d, a, in13, S44, 1309151649); /* 60 */
  III ( a, b, c, d, in4, S41, 4149444226); /* 61 */
  III ( d, a, b, c, in11, S42, 3174756917); /* 62 */
  III ( c, d, a, b, in2, S43,  718787259); /* 63 */
  III ( b, c, d, a, in9, S44, 3951481745); /* 64 */

  a += a0;
  b += b0;
  c += c0;
  d += d0;

  *a1 = a;
  *b1 = b;
  *c1 = c;
  *d1 = d;
}

__device__ void Encode( unsigned char *output, uint32_t *input, unsigned int len )
{
	unsigned int i, j;

	for (i = 0, j = 0; j < len; i++, j += 4) {
	  output[j] = (unsigned char)(input[i] & 0xff);
	  output[j+1] = (unsigned char)((input[i] >> 8) & 0xff);
	  output[j+2] = (unsigned char)((input[i] >> 16) & 0xff);
	  output[j+3] = (unsigned char)((input[i] >> 24) & 0xff);
	}
}

static void Decode( uint32_t *output, unsigned char *input, unsigned int len )
{
	unsigned int i, j;

	for (i = 0, j = 0; j < len; i++, j += 4)
		output[i] = ((uint32_t)input[j]) | (((uint32_t)input[j+1]) << 8) | (((uint32_t)input[j+2]) << 16) | (((uint32_t)input[j+3]) << 24);
}
