#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

/*
Author: Andrew DiPrinzio 
Course: EN605.417.FA
*/

// Structure that holds program arguments specifying number of threads/blocks
// to use.
typedef struct {    
    uint32_t num_threads;
    uint32_t block_size;
} Arguments;

// Parse the command line arguments using getopt and return an Argument structure
// GetOpt requies the POSIX C Library
static Arguments parse_arguments(const int argc, char ** argv){   
    // Argument format string for getopt
    static const char * _ARG_STR = "ht:b:";
    // Initialize arguments to their default values    
    Arguments args;    
    args.num_threads = DEFAULT_NUM_THREADS;    
    args.block_size = DEFAULT_BLOCK_SIZE;
    // Parse any command line options
    int c;
    int value;
    while ((c = getopt(argc, argv, _ARG_STR)) != -1) {
        switch (c) {
            case 't':
                value = atoi(optarg);
                args.num_threads = value;
                break;
            case 'b':
                // Normal argument
                value = atoi(optarg);
                args.block_size = value;
                break;
            case 'h':
                // 'help': print usage, then exit
                // note the fall through
                usage();
            default:
                exit(-1);
        }
    }
    return args;
}

//Kernel that adds two vectors
__global__
void add_ab(int *a, const int *b)
{
	const unsigned int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	a[thread_idx] += b[thread_idx];
}

// Helper function to generate a random number within a defined range
int random(int min, int max){
    return min + rand() / (RAND_MAX / (max - min + 1) + 1);
}

void run_vector_add(int * num_threads, int * block_size)
{ 
    printf("Running random vector add with %d threads and a block size of %d\n",*num_threads,*num_threads);
    int array_size = *num_threads;
    int array_size_in_bytes = (sizeof(int) * (array_size));

    /* Randomly generate input vectors and dynamically allocate their memory */
    int * a; 
    int * b;
    
    a = (int*)malloc(array_size * sizeof(int));
    b = (int*)malloc(array_size * sizeof(int));

    int i;
    for (i = 0; i < array_size; i++) {
        a[i] = random(0,100);
    }
    for (i = 0; i < array_size; i++) {
        b[i] = random(0,100);
    }

	/* Declare pointers for GPU based params */
    int *a_d;
	int *b_d;

	hipMalloc((void**)&a_d, array_size_in_bytes);
	hipMalloc((void**)&b_d, array_size_in_bytes);
	hipMemcpy( a_d, a, array_size_in_bytes, hipMemcpyHostToDevice );
	hipMemcpy( b_d, b, array_size_in_bytes, hipMemcpyHostToDevice );

	const unsigned int num_blocks = array_size / *block_size;
	const unsigned int num_threads_per_blk = array_size/num_blocks;

	/* Execute our kernel */
	add_ab<<<num_blocks, num_threads_per_blk>>>(a_d, b_d);

	/* Free the arrays on the GPU as now we're done with them */
	hipMemcpy(a, a_d, array_size_in_bytes, hipMemcpyDeviceToHost );
	hipFree(a_d);
	hipFree(b_d);

	/* Iterate through the result array and print */
    for(unsigned int i = 0; i < array_size; i++)
	{
		printf("Sum #%d: %d\n",i,a[i]);
	}
}

int main(int argc, char ** argv)
{
    Arguments args = parse_arguments(argc, argv);
    printf("Num Threads: %u, Block Size: %u\n", args.num_threads, args.block_size);

    run_vector_add(args.num_threads, args.block_size);
    
	return EXIT_SUCCESS;
}